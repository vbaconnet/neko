
#include <hip/hip_runtime.h>
/**
 * Device kernel for scalar apply for a Dirichlet condition
 */
__global__ void dirichlet_apply_scalar_kernel(const int * __restrict__ msk,
					      double * __restrict__ x,
					      const double g,
					      const int m) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = (idx + 1); i < m; i += str) {
    const int k = msk[i] -1;
    x[k] = g;
  }
}

/**
 * Device kernel for vector apply for a Dirichlet condition
 */
__global__ void dirichlet_apply_vector_kernel(const int * __restrict__ msk,
					      double * __restrict__ x,
					      double * __restrict__ y,
					      double * __restrict__ z,
					      const double g,
					      const int m) {

  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const int str = blockDim.x * gridDim.x;

  for (int i = (idx + 1); i < m; i += str) {
    const int k = msk[i] -1;
    x[k] = g;
    y[k] = g;
    z[k] = g;
  }
}

extern "C" {

  /** 
   * Fortran wrapper for device dirichlet apply scalar
   */
  void cuda_dirichlet_apply_scalar(void *msk, void *x,
				  double *g, int *m) {
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*m)+1024 - 1)/ 1024, 1, 1);

    dirichlet_apply_scalar_kernel<<<nblcks, nthrds>>>((int *) msk,
						      (double *) x, *g, *m);
  }
  
  /** 
   * Fortran wrapper for device dirichlet apply vector
   */
  void cuda_dirichlet_apply_vector(void *msk, void *x, void *y,
				  void *z, double *g, int *m) {
    
    const dim3 nthrds(1024, 1, 1);
    const dim3 nblcks(((*m)+1024 - 1)/ 1024, 1, 1);

    dirichlet_apply_vector_kernel<<<nblcks, nthrds>>>((int *) msk,
						      (double *) x,
						      (double *) y,
						      (double *) z,
						      *g, *m);
  }
 
}
